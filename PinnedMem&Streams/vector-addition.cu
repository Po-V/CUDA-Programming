// This program computes the sum of 2 arrays on the GPU using CUDA with pinned memory and streams

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


// CUDA kernel function for vector addition
__global__ void vecAdd_kernel(float *a, float *b, float *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        c[i] = a[i] + b[i];
}

// GPU function to set up and launch the kernel
void vecAdd_gpu(float *a, float *b, float *c, int n)
{
    float *a_d, *b_d, *c_d;
    int size = n * sizeof(float);

    // Allocate device memory
    hipMalloc((void **)&a_d, size);
    hipMalloc((void **)&b_d, size);
    hipMalloc((void **)&c_d, size);
    hipDeviceSynchronize();

    unsigned int numStreams = 10; // divide input into 32 segments
    hipStream_t stream[numStreams];
    for(unsigned int s = 0; s < numStreams; ++s){
        // create a stream
        hipStreamCreate(&stream[s]);
    }


    // stream the segments
    unsigned int numSegments = numStreams;
    unsigned int segmentSize = (n + numSegments -1) / numSegments;

    // determine input for each segment
    for(unsigned int s = 0; s < numSegments; ++s){

        // finding the segment bounds
        unsigned int start = s*segmentSize;
        // finding end of segment and check if end execeed boundary
        unsigned int end = (start + segmentSize < n)?(start + segmentSize):n;
        // number of elements in a segment
        unsigned int Nsegment = end - start;

        // Copy input data from host to device. copy from offset of start
        hipMemcpyAsync(&a_d[start], &a[start], Nsegment*sizeof(float), hipMemcpyHostToDevice, stream[s]);
        hipMemcpyAsync(&b_d[start], &b[start], Nsegment*sizeof(float), hipMemcpyHostToDevice, stream[s]);

        // Launch the kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
        // kernel calls by definition is asynchronous. Kernel would not start before cudaMemcpyAsync completed
        vecAdd_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream[s]>>>(&a_d[start], &b_d[start], &c_d[start], Nsegment);

        
        // Copy result back to host
        hipMemcpyAsync(&c[start], &c_d[start], Nsegment*sizeof(float), hipMemcpyDeviceToHost, stream[s]);
    }
    hipDeviceSynchronize();

    // Free device memory
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    hipDeviceSynchronize();

}

// Function to initialize array with random float values
void initializeRandomArray(float *arr, int size)
{
    for (int i = 0; i < size; i++)
    {
        arr[i] = (float)rand() / RAND_MAX * 10.0f;  // Random float between 0 and 10
    }
}

// Initialize an array of size "N" with numbers between 0 and 100
void init_array(float *a, int N){
    for(int i = 0; i < N; i++){
        a[i] = (float) rand() / RAND_MAX * 10.0f;
    }
}


// Main function
int main()
{
    srand(time(NULL));  // Seed for random number generation
    int N = 10;

    // Allocate memory for vectors
    float *a ; hipHostMalloc((void**) &a, N * sizeof(float), hipHostMallocDefault);
    float *b ; hipHostMalloc((void**) &b, N * sizeof(float), hipHostMallocDefault);
    float *c ; hipHostMalloc((void**) &c, N * sizeof(float), hipHostMallocDefault);

    // Initialize input vectors with random values
    init_array(a, N);
    init_array(b, N);

    // call kernel
    vecAdd_gpu(a, b, c, N);

    // Free host memory
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);

    return 0;
}