#include "hip/hip_runtime.h"
#include "common.h"
#define LOCAL_QUEUE_SIZE 2048 // total threads

// use csr representation for vertex centric approach
__global__ void bfs_kernel(CSRGraph csrGraph, unsigned int* level, unsigned int* prevFrontier, unsigned int* currFrontier, unsigned int numPrevFrontier, unsigned int* numCurrFrontier, unsigned int currLevel){
    // level = level of each vertex in the graph
    // prevFrontier = array that contains indices of previous frontier. Initialized to source vertex
    // numCurrFrontier = number of vetices in the current frontier. INtialized to 0
    // currLevel is the level we are currently in

    __shared__ unsigned int currFrontier_s[LOCAL_QUEUE_SIZE]; // create local queue shared memory
    __shared__ unsigned int numCurrFrontier_s;
    if(threadIdx.x == 0){
        numCurrFrontier_s = 0;
    }
    __syncthreads();
    
    // used threads to index previous frontier
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    // boundaary check with number of previous frontier
    if( i < numPrevFrontier){
        // threads access vertex it is responsible for
        unsigned int vertex = prevFrontier[i];
        for(unsigned int edge = csrGraph.srcPtrs[vertex]; edge < csrGraph.srcPtrs[vertex + 1]; ++edge){
            // find index of neighbour at destination of edge
            unsigned int neighbor = csrGraph.dst[edge];
            // check if neighbor is visited
            // if level of neigbor is UINT_MAX, swap it with current level
            // add neighbor to the current frontier at the index. Multiple threads in the previous frontier may have same neighbor
            // may have same neighbor added to current frontier multiple times. 
            // use atomic operation (returns old value) to ensure only 1 thread does the comparison at a time
            if(atomicCAS(&level[neighbor], UINT_MAX, currLevel) == UINT_MAX){
                // increment index of current frontier. Use atomic add to prevent race condition
                //unsigned int currFrontierIdx = atomicAdd(numCurrFrontier, 1);
                //currFrontier[currFrontierIdx] = neighbor;

                // add to local frontier
                unsigned int currFrontierIdx_s = atomicAdd(&numCurrFrontier_s, 1);

                // check for overflow
                if(currFrontierIdx_s < LOCAL_QUEUE_SIZE){
                    
                    // insert to current frontier
                    currFrontier_s[currFrontierIdx_s] = neighbor;
                }else{
                    // if overflow
                    numCurrFrontier_s = LOCAL_QUEUE_SIZE;
                    // add to global queue
                    unsigned int currFrontierIdx = atomicAdd(numCurrFrontier, 1);
                    currFrontier[currFrontierIdx] = neighbor;
                }
                
            }
        }
    }
    __syncthreads();

    // increment global queue counter by number of local elements
    __shared__ unsigned int currFrontierStartIdx;
    // only 1 thread writes to global counter 
    if(threadIdx.x == 0){
        currFrontierStartIdx = atomicAdd(numCurrFrontier, numCurrFrontier_s);
    }
    __syncthreads(); // all threads wait for thread 0 to allocate space in global queue

    // since queue size is 2048 and maximum number of threads may be 255
    for(unsigned int currFrontierIdx_s =threadIdx.x; currFrontierIdx_s< numCurrFrontier_s; currFrontierIdx_s += blockDim.x){
        // thread 0 will write to index 0 at global memory and so on
        currFrontier[currFrontierStartIdx + currFrontierIdx_s] = currFrontier_s[currFrontierIdx_s];
    }

}

void bfs_gpu(CSRGraph csrGraph, unsigned int srcVertex, unsigned int* level){

    // allocate GPU memory
    CSRGraph csrGraph_d;
    csrGraph_d.numVertices = csrGraph.numVertices;
    csrGraph_d.numEdges = csrGraph.numEdges;
    hipMalloc((void**) &csrGraph_d.srcPtrs, (csrGraph_d.numVertices +1)*sizeof(unsigned int));
    hipMalloc((void**) &csrGraph_d.dst, csrGraph_d.numEdges*sizeof(unsigned int));
    unsigned int* level_d;
    hipMalloc((void**) &level_d, csrGraph_d.numVertices*sizeof(unsigned int));
    unsigned int* buffer1_d;
    unsigned int* buffer2_d;
    hipMalloc((void**) &buffer1_d, csrGraph_d.numVertices*sizeof(unsigned int));
    hipMalloc((void**) &buffer2_d, csrGraph_d.numVertices*sizeof(unsigned int));
    unsigned int* numCurrFrontier_d;
    hipMalloc((void**) &numCurrFrontier_d, sizeof(unsigned int));
    unsigned int* prevFrontier_d = buffer1_d;
    unsigned int* currFrontier_d = buffer2_d;
    hipDeviceSynchronize();
    
    // copy data to GPU
    hipMemcpy(csrGraph_d.srcPtrs, csrGraph.srcPtrs, (csrGraph_d.numVertices +1)*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(csrGraph_d.dst, csrGraph.dst, csrGraph_d.numEdges*sizeof(unsigned int), hipMemcpyHostToDevice);
    level[srcVertex] = 0;
    hipMemcpy(level_d, level, csrGraph_d.numVertices*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(prevFrontier_d, &srcVertex, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // call kernel
    unsigned int numPrevFrontier = 1;
    unsigned int numThreadsPerBlock = 256;
    for(unsigned int currLevel = 1; numPrevFrontier > 0; ++currLevel){
        
        // visit vertices in previous frontier
        hipMemset(numCurrFrontier_d, 0, sizeof(unsigned int));
        unsigned int numBLocks = (numPrevFrontier + numThreadsPerBlock - 1)/numThreadsPerBlock;
        bfs_kernel<<<numBLocks, numThreadsPerBlock>>>(csrGraph_d, level_d, prevFrontier_d, currFrontier_d, numPrevFrontier, numCurrFrontier_d, currLevel);

        // swap buffers
        unsigned int* tmp = prevFrontier_d;
        prevFrontier_d = currFrontier_d;
        currFrontier_d = tmp;
        hipMemcpy(&numPrevFrontier, numCurrFrontier_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
    }
    hipDeviceSynchronize();

    // copy data from GPU
    hipMemcpy(level, level_d, csrGraph_d.numVertices*sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // free GPU memory
    hipFree(csrGraph_d.srcPtrs);
    hipFree(csrGraph_d.dst);
    hipFree(level_d);
    hipFree(buffer1_d);
    hipFree(buffer2_d);
    hipFree(numCurrFrontier_d);
    hipDeviceSynchronize();
}

int main(){

    // example graph in CSR format
    CSRGraph graph;
    graph.numVertices = 4;
    graph.numEdges = 8;

    int adjacencyMatrix[graph.numVertices][graph.numVertices] = {
        {0, 1, 1, 0},
        {1, 0, 1, 1},
        {1, 1, 0, 0},
        {0, 1, 0, 0}
    };


    graph.srcPtrs = (unsigned int* )malloc((graph.numVertices + 1)*sizeof(unsigned int));
    graph.dst = (unsigned int*)malloc(graph.numEdges*sizeof(unsigned int));

    // Fill in the CSR data using loops
    unsigned int edgeCount = 0;
    for (int i = 0; i < graph.numVertices; i++) {
        graph.srcPtrs[i] = edgeCount;
        for (int j = 0; j < graph.numVertices; j++) {
            if (adjacencyMatrix[i][j] == 1) {
                graph.dst[edgeCount] = j;
                edgeCount++;
            }
        }
    }
    graph.srcPtrs[graph.numVertices] = edgeCount;

    // Allocate and initialize level array. Level is distance from source vertex
    unsigned int* level = (unsigned int*)malloc(graph.numVertices * sizeof(unsigned int));
    for (unsigned int i = 0; i < graph.numVertices; i++) {
        level[i] = UINT_MAX;
    }

    // choose a source vertex
    unsigned int srcVertex = 0;

    // run BFS on GPU
    bfs_gpu(graph, srcVertex, level);

    free(graph.srcPtrs);
    free(graph.dst);
    free(level);

    return 0;
}

